#include "hip/hip_runtime.h"
#include "plssvm/backends/CUDA/cuda-kernel.cuh"

namespace plssvm {

template <typename real_type>
__global__ void kernel_q_linear(real_type *q, const real_type *data_d, const real_type *datlast, const int Nrows, const int start, const int end) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    real_type temp{ 0.0 };
    for (int i = start; i < end; ++i) {
        temp += data_d[i * Nrows + index] * datlast[i];
    }
    q[index] = temp;
}
template __global__ void kernel_q_linear(float *, const float *, const float *, const int, const int, const int);
template __global__ void kernel_q_linear(double *, const double *, const double *, const int, const int, const int);

template <typename real_type>
__global__ void kernel_q_poly(real_type *q, const real_type *data_d, const real_type *datlast, const int Nrows, const int Ncols, const real_type degree, const real_type gamma, const real_type coef0) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    real_type temp{ 0.0 };
    for (int i = 0; i < Ncols; ++i) {
        temp += data_d[i * Nrows + index] * datlast[i];
    }
    q[index] = pow(gamma * temp + coef0, degree);
}
template __global__ void kernel_q_poly(float *, const float *, const float *, const int, const int, const float, const float, const float);
template __global__ void kernel_q_poly(double *, const double *, const double *, const int, const int, const double, const double, const double);

template <typename real_type>
__global__ void kernel_q_radial(real_type *q, const real_type *data_d, const real_type *datlast, const int Nrows, const int Ncols, const real_type gamma) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    real_type temp{ 0.0 };
    for (int i = 0; i < Ncols; ++i) {
        temp += (data_d[i * Nrows + index] - datlast[i]) * (data_d[i * Nrows + index] - datlast[i]);
    }
    q[index] = exp(-gamma * temp);
}
template __global__ void kernel_q_radial(float *, const float *, const float *, const int, const int, const float);
template __global__ void kernel_q_radial(double *, const double *, const double *, const int, const int, const double);

}  // namespace plssvm