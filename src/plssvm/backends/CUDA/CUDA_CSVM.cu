#include "hip/hip_runtime.h"
#include "plssvm/backends/CUDA/CUDA_CSVM.hpp"
#include "plssvm/backends/CUDA/cuda-kernel.cuh"
#include "plssvm/backends/CUDA/cuda-kernel.hpp"
#include "plssvm/backends/CUDA/svm-kernel.cuh"
#include "plssvm/detail/operators.hpp"

#include <chrono>

namespace plssvm {

int CUDADEVICE = 0;

#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s:%d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

int count_devices = 1;

template <typename T>
CUDA_CSVM<T>::CUDA_CSVM(parameter<T> &params) :
    CUDA_CSVM{ params.kernel, params.degree, params.gamma, params.coef0, params.cost, params.epsilon, params.print_info } {}

template <typename T>
CUDA_CSVM<T>::CUDA_CSVM(kernel_type kernel, real_type degree, real_type gamma, real_type coef0, real_type cost, real_type epsilon, bool print_info) :
    CSVM<T>{ kernel, degree, gamma, coef0, cost, epsilon, print_info } {
    gpuErrchk(hipGetDeviceCount(&count_devices));
    datlast_d = std::vector<real_type *>(count_devices);
    data_d = std::vector<real_type *>(count_devices);

    std::cout << "GPUs found: " << count_devices << std::endl;
}

template <typename T>
void CUDA_CSVM<T>::setup_data_on_device() {
    for (size_type device = 0; device < count_devices; ++device) {
        gpuErrchk(hipSetDevice(device));
        gpuErrchk(hipMalloc((void **) &datlast_d[device],
                             (num_data_points_ - 1 + THREADBLOCK_SIZE * INTERNALBLOCK_SIZE) * sizeof(real_type)));
    }
    std::vector<real_type> datalast(data_[num_data_points_ - 1]);
    datalast.resize(num_data_points_ - 1 + THREADBLOCK_SIZE * INTERNALBLOCK_SIZE);
    #pragma omp parallel for
    for (size_type device = 0; device < count_devices; ++device) {
        gpuErrchk(hipSetDevice(device));
        gpuErrchk(hipMemcpy(datlast_d[device], datalast.data(), (num_data_points_ - 1 + THREADBLOCK_SIZE * INTERNALBLOCK_SIZE) * sizeof(real_type), hipMemcpyHostToDevice));
    }
    datalast.resize(num_data_points_ - 1);
    for (size_type device = 0; device < count_devices; ++device) {
        gpuErrchk(hipSetDevice(device));
        gpuErrchk(hipMalloc((void **) &data_d[device],
                             num_features_ * (num_data_points_ + THREADBLOCK_SIZE * INTERNALBLOCK_SIZE) * sizeof(real_type)));
    }

    auto begin_transform = std::chrono::high_resolution_clock::now();
    const std::vector<real_type> transformet_data = base_type::transform_data(THREADBLOCK_SIZE * INTERNALBLOCK_SIZE);
    auto end_transform = std::chrono::high_resolution_clock::now();
    if (print_info_) {
        std::clog << std::endl
                  << data_.size() << " Datenpunkte mit Dimension " << num_features_ << " in "
                  << std::chrono::duration_cast<std::chrono::milliseconds>(end_transform - begin_transform).count()
                  << " ms transformiert" << std::endl;
    }
    #pragma omp parallel for
    for (size_type device = 0; device < count_devices; ++device) {
        gpuErrchk(hipSetDevice(device));

        gpuErrchk(hipMemcpy(data_d[device], transformet_data.data(), num_features_ * (num_data_points_ - 1 + THREADBLOCK_SIZE * INTERNALBLOCK_SIZE) * sizeof(real_type), hipMemcpyHostToDevice));
    }
}

template <typename T>
auto CUDA_CSVM<T>::generate_q() -> std::vector<real_type> {
    if (print_info_) {
        std::cout << "kernel_q" << std::endl;
    }

    const size_type dept = num_data_points_ - 1;
    const size_type boundary_size = THREADBLOCK_SIZE * INTERNALBLOCK_SIZE;
    const size_type dept_all = dept + boundary_size;
    const int Ncols = num_features_;
    const int Nrows = dept + THREADBLOCK_SIZE * INTERNALBLOCK_SIZE;

    std::vector<real_type *> q_d(count_devices);
    for (size_type device = 0; device < count_devices; ++device) {
        gpuErrchk(hipSetDevice(device));
        gpuErrchk(hipMalloc((void **) &q_d[device], dept_all * sizeof(real_type)));
        gpuErrchk(hipMemset(q_d[device], 0, dept_all * sizeof(real_type)));
    }
    gpuErrchk(hipDeviceSynchronize());
    for (size_type device = 0; device < count_devices; ++device) {
        gpuErrchk(hipSetDevice(device));

        const int start = device * Ncols / count_devices;
        const int end = (device + 1) * Ncols / count_devices;
        kernel_q<<<((int) dept / CUDABLOCK_SIZE) + 1, std::min((size_type) CUDABLOCK_SIZE, dept)>>>(q_d[device],
                                                                                                    data_d[device],
                                                                                                    datlast_d[device],
                                                                                                    Nrows,
                                                                                                    start,
                                                                                                    end);
        gpuErrchk(hipPeekAtLastError());
    }
    gpuErrchk(hipDeviceSynchronize());

    std::vector<real_type> q(dept);
    gpuErrchk(hipSetDevice(0));
    gpuErrchk(hipMemcpy(q.data(), q_d[0], dept * sizeof(real_type), hipMemcpyDeviceToHost));
    std::vector<real_type> ret(dept_all);
    for (size_type device = 1; device < count_devices; ++device) {
        gpuErrchk(hipSetDevice(device));
        gpuErrchk(hipMemcpy(ret.data(), q_d[device], dept * sizeof(real_type), hipMemcpyDeviceToHost));
        for (size_type i = 0; i < dept; ++i) {
            q[i] += ret[i];
        }
    }
    return q;
}

template <typename T>
auto CUDA_CSVM<T>::solver_CG(const std::vector<real_type> &b, const size_type imax, const real_type eps, const std::vector<real_type> &q) -> std::vector<real_type> {
    const size_type dept = num_data_points_ - 1;
    const size_type boundary_size = THREADBLOCK_SIZE * INTERNALBLOCK_SIZE;
    const size_type dept_all = dept + boundary_size;
    std::vector<real_type> zeros(dept_all, 0.0);

    // dim3 grid((int)dept/(CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) + 1,(int)dept/(CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) + 1);
    dim3 block(THREADBLOCK_SIZE, THREADBLOCK_SIZE);

    real_type *d;
    std::vector<real_type> x(dept_all, 1.0);
    std::fill(x.end() - boundary_size, x.end(), 0.0);

    std::vector<real_type *> x_d(count_devices);
    std::vector<real_type> r(dept_all, 0.0);
    std::vector<real_type *> r_d(count_devices);
    for (size_type device = 0; device < count_devices; ++device) {
        gpuErrchk(hipSetDevice(device));
        gpuErrchk(hipMalloc((void **) &x_d[device], dept_all * sizeof(real_type)));
        gpuErrchk(hipMemcpy(x_d[device], x.data(), dept_all * sizeof(real_type), hipMemcpyHostToDevice));
        gpuErrchk(hipMalloc((void **) &r_d[device], dept_all * sizeof(real_type)));
    }

    gpuErrchk(hipSetDevice(0));
    gpuErrchk(hipMemcpy(r_d[0], b.data(), dept * sizeof(real_type), hipMemcpyHostToDevice));
    gpuErrchk(hipMemset(r_d[0] + dept, 0, (dept_all - dept) * sizeof(real_type)));
    #pragma omp parallel for
    for (size_type device = 1; device < count_devices; ++device) {
        gpuErrchk(hipSetDevice(device));
        gpuErrchk(hipMemset(r_d[device], 0, dept_all * sizeof(real_type)));
    }
    d = new real_type[dept];

    const int Ncols = num_features_;
    const int Nrows = dept + THREADBLOCK_SIZE * INTERNALBLOCK_SIZE;
    gpuErrchk(hipDeviceSynchronize());

    std::vector<real_type *> q_d(count_devices);
    for (size_type device = 0; device < count_devices; ++device) {
        gpuErrchk(hipSetDevice(device));
        gpuErrchk(hipMalloc((void **) &q_d[device], dept_all * sizeof(real_type)));
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipMemset(q_d[device], 0, dept_all * sizeof(real_type)));
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipMemcpy(q_d[device], q.data(), dept_all * sizeof(real_type), hipMemcpyHostToDevice));
    }

    gpuErrchk(hipDeviceSynchronize());

    switch (kernel_) {
        case kernel_type::linear: {
            #pragma omp parallel for
            for (size_type device = 0; device < count_devices; ++device) {
                gpuErrchk(hipSetDevice(device));
                dim3 grid(static_cast<size_type>(ceil(
                              static_cast<real_type>(dept) / static_cast<real_type>(THREADBLOCK_SIZE * INTERNALBLOCK_SIZE))),
                          static_cast<size_type>(ceil(static_cast<real_type>(dept) / static_cast<real_type>(THREADBLOCK_SIZE * INTERNALBLOCK_SIZE))));
                const int start = device * Ncols / count_devices;
                const int end = (device + 1) * Ncols / count_devices;
                kernel_linear<<<grid, block>>>(q_d[device], r_d[device], x_d[device], data_d[device], QA_cost_, 1 / cost_, Ncols, Nrows, -1, start, end);
                gpuErrchk(hipPeekAtLastError());
            }
            break;
        }
        case kernel_type::polynomial:
            // kernel_poly<<<grid,block>>>(q_d, r_d, x_d,data_d, QA_cost_, 1/cost, num_features_ , dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD), -1, gamma_, coef0_, degree_);
            break;
        case kernel_type::rbf:
            // kernel_radial<<<grid,block>>>(q_d, r_d, x_d,data_d, QA_cost_, 1/cost_, num_features_ , dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD), -1, gamma_);
            break;
        default:
            throw std::runtime_error("Can not decide which kernel!");
    }

    // hipMemcpy(r, r_d, dept*sizeof(real_type), hipMemcpyDeviceToHost);
    gpuErrchk(hipDeviceSynchronize());
    {
        gpuErrchk(hipSetDevice(0));
        gpuErrchk(hipMemcpy(r.data(), r_d[0], dept_all * sizeof(real_type), hipMemcpyDeviceToHost));
        for (int device = 1; device < count_devices; ++device) {
            gpuErrchk(hipSetDevice(device));
            std::vector<real_type> ret(dept_all);
            gpuErrchk(hipMemcpy(ret.data(), r_d[device], dept_all * sizeof(real_type), hipMemcpyDeviceToHost));
            for (size_type j = 0; j <= dept; ++j) {
                r[j] += ret[j];
            }
        }
    }
    real_type delta = mult(r.data(), r.data(), dept);  // TODO:
    const real_type delta0 = delta;
    real_type alpha_cd, beta;
    std::vector<real_type> Ad(dept);

    std::vector<real_type *> Ad_d(count_devices);
    for (size_type device = 0; device < count_devices; ++device) {
        gpuErrchk(hipSetDevice(device));
        gpuErrchk(hipMalloc((void **) &Ad_d[device], dept_all * sizeof(real_type)));
        gpuErrchk(hipMemcpy(r_d[device], r.data(), dept_all * sizeof(real_type), hipMemcpyHostToDevice));
    }
    //hipHostMalloc((void **) &Ad, dept *sizeof(real_type));

    size_type run;
    for (run = 0; run < imax; ++run) {
        if (print_info_) {
            std::cout << "Start Iteration: " << run << std::endl;
        }
        //Ad = A * d
        for (size_type device = 0; device < count_devices; ++device) {
            gpuErrchk(hipSetDevice(device));
            gpuErrchk(hipMemset(Ad_d[device], 0, dept_all * sizeof(real_type)));
            gpuErrchk(hipMemset(r_d[device] + dept, 0, (dept_all - dept) * sizeof(real_type)));
        }
        switch (kernel_) {
            case kernel_type::linear: {
                #pragma omp parallel for
                for (size_type device = 0; device < count_devices; ++device) {
                    gpuErrchk(hipSetDevice(device));
                    dim3 grid(static_cast<size_type>(ceil(static_cast<real_type>(dept) / static_cast<real_type>(THREADBLOCK_SIZE * INTERNALBLOCK_SIZE))),
                              static_cast<size_type>(ceil(static_cast<real_type>(dept) / static_cast<real_type>(THREADBLOCK_SIZE * INTERNALBLOCK_SIZE))));
                    const int start = device * Ncols / count_devices;
                    const int end = (device + 1) * Ncols / count_devices;
                    kernel_linear<<<grid, block>>>(q_d[device], Ad_d[device], r_d[device], data_d[device], QA_cost_, 1 / cost_, Ncols, Nrows, 1, start, end);
                    gpuErrchk(hipPeekAtLastError());
                }
            } break;
            case kernel_type::polynomial:
                // kernel_poly<<<grid,block>>>(q_d, Ad_d, r_d, data_d, QA_cost_, 1/cost_, num_features_, dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) , 1, gamma_, coef0_, degree_);
                break;
            case kernel_type::rbf:
                // kernel_radial<<<grid,block>>>(q_d, Ad_d, r_d, data_d, QA_cost_, 1/cost_, num_features_, dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD), 1, gamma_);
                break;
            default:
                throw std::runtime_error("Can not decide which kernel!");
        }

        for (size_type i = 0; i < dept; ++i) {
            d[i] = r[i];
        }

        gpuErrchk(hipDeviceSynchronize());
        {
            std::vector<real_type> buffer(dept_all, 0);
            for (size_type device = 0; device < count_devices; ++device) {
                gpuErrchk(hipSetDevice(device));
                std::vector<real_type> ret(dept_all, 0);
                gpuErrchk(hipMemcpy(ret.data(), Ad_d[device], dept_all * sizeof(real_type), hipMemcpyDeviceToHost));
                for (size_type j = 0; j <= dept; ++j) {
                    buffer[j] += ret[j];
                }
            }
            std::copy(buffer.begin(), buffer.begin() + dept, Ad.data());
            for (size_type device = 0; device < count_devices; ++device) {
                gpuErrchk(hipSetDevice(device));
                gpuErrchk(
                    hipMemcpy(Ad_d[device], buffer.data(), dept_all * sizeof(real_type), hipMemcpyHostToDevice));
            }
        }

        alpha_cd = delta / mult(d, Ad.data(), dept);
        // add_mult<<< ((int) dept/1024) + 1, std::min(1024, dept)>>>(x_d,r_d,alpha_cd,dept);
        //TODO: auf GPU
        std::vector<real_type> buffer_r(dept_all);
        hipSetDevice(0);
        gpuErrchk(hipMemcpy(buffer_r.data(), r_d[0], dept_all * sizeof(real_type), hipMemcpyDeviceToHost));
        add_mult_(((int) dept / 1024) + 1, std::min(1024, (int) dept), x.data(), buffer_r.data(), alpha_cd, dept);

        #pragma omp parallel for
        for (size_type device = 0; device < count_devices; ++device) {
            gpuErrchk(hipSetDevice(device));
            gpuErrchk(hipMemcpy(x_d[device], x.data(), dept_all * sizeof(real_type), hipMemcpyHostToDevice));
        }
        if (run % 50 == 49) {
            std::vector<real_type> buffer(b);
            buffer.resize(dept_all);
            gpuErrchk(hipSetDevice(0));
            gpuErrchk(hipMemcpy(r_d[0], buffer.data(), dept_all * sizeof(real_type), hipMemcpyHostToDevice));
            #pragma omp parallel for
            for (size_type device = 1; device < count_devices; ++device) {
                gpuErrchk(hipSetDevice(device));
                gpuErrchk(hipMemset(r_d[device], 0, dept_all * sizeof(real_type)));
            }
            switch (kernel_) {
                case kernel_type::linear: {
                    #pragma omp parallel for
                    for (size_type device = 0; device < count_devices; ++device) {
                        gpuErrchk(hipSetDevice(device));
                        const int start = device * Ncols / count_devices;
                        const int end = (device + 1) * Ncols / count_devices;
                        dim3 grid(static_cast<size_type>(ceil(static_cast<real_type>(dept) / static_cast<real_type>(THREADBLOCK_SIZE * INTERNALBLOCK_SIZE))),
                                  static_cast<size_type>(ceil(static_cast<real_type>(dept) / static_cast<real_type>(THREADBLOCK_SIZE * INTERNALBLOCK_SIZE))));
                        kernel_linear<<<grid, block>>>(q_d[device], r_d[device], x_d[device], data_d[device], QA_cost_, 1 / cost_, Ncols, Nrows, -1, start, end);
                        gpuErrchk(hipPeekAtLastError());
                    }
                } break;
                case kernel_type::polynomial:
                    // kernel_poly<<<grid,block>>>(q_d, r_d, x_d, data_d, QA_cost_, 1/cost_, num_features_, dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD), -1, gamma_, coef0_, degree_);
                    break;
                case kernel_type::rbf:
                    // kernel_radial<<<grid,block>>>(q_d, r_d, x_d, data_d, QA_cost_, 1/cost_, num_features_, dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) , -1, gamma_);
                    break;
                default:
                    throw std::runtime_error("Can not decide wich kernel!");
            }
            gpuErrchk(hipDeviceSynchronize());
            // hipMemcpy(r, r_d, dept*sizeof(real_type), hipMemcpyDeviceToHost);

            {
                gpuErrchk(hipSetDevice(0));
                gpuErrchk(hipMemcpy(r.data(), r_d[0], dept_all * sizeof(real_type), hipMemcpyDeviceToHost));
                #pragma omp parallel for
                for (size_type device = 1; device < count_devices; ++device) {
                    gpuErrchk(hipSetDevice(device));
                    std::vector<real_type> ret(dept_all, 0);
                    gpuErrchk(
                        hipMemcpy(ret.data(), r_d[device], dept_all * sizeof(real_type), hipMemcpyDeviceToHost));
                    for (size_type j = 0; j <= dept; ++j) {
                        r[j] += ret[j];
                    }
                }
                #pragma omp parallel for
                for (size_type device = 0; device < count_devices; ++device) {
                    gpuErrchk(hipSetDevice(device));
                    gpuErrchk(hipMemcpy(r_d[device], r.data(), dept_all * sizeof(real_type), hipMemcpyHostToDevice));
                }
            }
        } else {
            for (size_type index = 0; index < dept; ++index) {
                r[index] -= alpha_cd * Ad[index];
            }
        }

        delta = mult(r.data(), r.data(), dept);  // TODO:
        if (delta < eps * eps * delta0) {
            break;
        }
        beta = -mult(r.data(), Ad.data(), dept) / mult(d, Ad.data(), dept);  // TODO:
        add(mult(beta, d, dept), r.data(), d, dept);                         // TODO:

        {
            std::vector<real_type> buffer(dept_all, 0.0);
            std::copy(d, d + dept, buffer.begin());
            #pragma omp parallel for
            for (size_type device = 0; device < count_devices; ++device) {
                gpuErrchk(hipSetDevice(device));
                gpuErrchk(
                    hipMemcpy(r_d[device], buffer.data(), dept_all * sizeof(real_type), hipMemcpyHostToDevice));
            }
        }
    }
    if (run == imax) {
        std::clog << "Regard reached maximum number of CG-iterations" << std::endl;
    }

    alpha_.resize(dept);
    std::vector<real_type> ret_q(dept);
    gpuErrchk(hipDeviceSynchronize());
    {
        std::vector<real_type> buffer(dept_all);
        std::copy(x.begin(), x.begin() + dept, alpha_.begin());
        gpuErrchk(hipSetDevice(0));
        gpuErrchk(hipMemcpy(buffer.data(), q_d[0], dept_all * sizeof(real_type), hipMemcpyDeviceToHost));
        std::copy(buffer.begin(), buffer.begin() + dept, ret_q.begin());
    }
    // hipMemcpy(&alpha[0],x_d, dept * sizeof(real_type), hipMemcpyDeviceToHost);
    // hipMemcpy(&ret_q[0],q_d, dept * sizeof(real_type), hipMemcpyDeviceToHost);
    // hipFree(Ad_d);
    // hipFree(r_d);
    // hipFree(datlast);
    // hipHostFree(Ad);
    // hipFree(x_d);
    // hipHostFree(r);
    // hipHostFree(d);
    return alpha_;
}

template class CUDA_CSVM<float>;
template class CUDA_CSVM<double>;

}  // namespace plssvm
