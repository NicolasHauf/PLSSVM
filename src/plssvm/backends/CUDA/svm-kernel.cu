#include "hip/hip_runtime.h"
#include "plssvm/backends/CUDA/svm-kernel.cuh"

#include "plssvm/backends/CUDA/detail/atomics.cuh"

#include "plssvm/typedef.hpp"

namespace plssvm {

template <typename real_type>
__global__ void kernel_linear(const real_type *q, real_type *ret, const real_type *d, const real_type *data_d, const real_type QA_cost, const real_type cost, const int Ncols, const int Nrows, const int add, const int start, const int end) {
    unsigned int i = blockIdx.x * blockDim.x * INTERNAL_BLOCK_SIZE;
    unsigned int j = blockIdx.y * blockDim.y * INTERNAL_BLOCK_SIZE;

    __shared__ real_type data_intern_i[THREAD_BLOCK_SIZE][INTERNAL_BLOCK_SIZE];
    __shared__ real_type data_intern_j[THREAD_BLOCK_SIZE][INTERNAL_BLOCK_SIZE];
    real_type matr[INTERNAL_BLOCK_SIZE][INTERNAL_BLOCK_SIZE] = { 0.0 };
    real_type data_j[INTERNAL_BLOCK_SIZE];

    if (i >= j) {
        i += threadIdx.x * INTERNAL_BLOCK_SIZE;
        const unsigned int ji = j + threadIdx.x * INTERNAL_BLOCK_SIZE;
        j += threadIdx.y * INTERNAL_BLOCK_SIZE;
        // cache data
        for (int vec_index = start * Nrows; vec_index < end * Nrows; vec_index += Nrows) {
            __syncthreads();
            #pragma unroll INTERNAL_BLOCK_SIZE
            for (std::size_t block_id = 0; block_id < INTERNAL_BLOCK_SIZE; ++block_id) {
                const size_t idx = block_id % THREAD_BLOCK_SIZE;
                if (threadIdx.y == idx) {
                    data_intern_i[threadIdx.x][block_id] = data_d[block_id + vec_index + i];
                }
                const size_t idx_2 = block_id + INTERNAL_BLOCK_SIZE % THREAD_BLOCK_SIZE;
                if (threadIdx.y == idx_2) {
                    data_intern_j[threadIdx.x][block_id] = data_d[block_id + vec_index + ji];
                }
            }
            __syncthreads();

            #pragma unroll INTERNAL_BLOCK_SIZE
            for (std::size_t data_index = 0; data_index < INTERNAL_BLOCK_SIZE; ++data_index) {
                data_j[data_index] = data_intern_j[threadIdx.y][data_index];
            }

            #pragma unroll INTERNAL_BLOCK_SIZE
            for (std::size_t l = 0; l < INTERNAL_BLOCK_SIZE; ++l) {
                const real_type data_i = data_intern_i[threadIdx.x][l];
                #pragma unroll INTERNAL_BLOCK_SIZE
                for (std::size_t k = 0; k < INTERNAL_BLOCK_SIZE; ++k) {
                    matr[k][l] += data_i * data_j[k];
                }
            }
        }

        #pragma unroll INTERNAL_BLOCK_SIZE
        for (std::size_t x = 0; x < INTERNAL_BLOCK_SIZE; ++x) {
            real_type ret_jx = 0.0;
            #pragma unroll INTERNAL_BLOCK_SIZE
            for (std::size_t y = 0; y < INTERNAL_BLOCK_SIZE; ++y) {
                real_type temp;
                if (start == 0) {
                    temp = (matr[x][y] + QA_cost - q[i + y] - q[j + x]) * add;
                } else {
                    temp = matr[x][y] * add;
                }
                if (i + x > j + y) {
                    // upper triangular matrix
                    atomicAdd(&ret[i + y], temp * d[j + x]);
                    ret_jx += temp * d[i + y];
                    // atomicAdd(&ret[j + x], temp * d[i + y]);
                } else if (i + x == j + y) {
                    // diagonal
                    if (start == 0) {
                        ret_jx += (temp + cost * add) * d[i + y];
                        // atomicAdd(&ret[j + x], (temp + cost * add) * d[i + y]);
                    } else {
                        ret_jx += temp * d[i + y];
                        // atomicAdd(&ret[j + x], temp * d[i + y]);
                    }
                }
            }
            atomicAdd(&ret[j + x], ret_jx);
        }
    }
}

template __global__ void kernel_linear(const float *, float *, const float *, const float *, const float, const float, const int, const int, const int, const int, const int);
template __global__ void kernel_linear(const double *, double *, const double *, const double *, const double, const double, const int, const int, const int, const int, const int);

template <typename real_type>  // TODO: remove start / end ?
__global__ void kernel_poly(const real_type *q, real_type *ret, const real_type *d, const real_type *data_d, const real_type QA_cost, const real_type cost, const int Ncols, const int Nrows, const int add, const int start, const int end, const real_type gamma, const real_type coef0, const real_type degree) {
    unsigned int i = blockIdx.x * blockDim.x * INTERNAL_BLOCK_SIZE;
    unsigned int j = blockIdx.y * blockDim.y * INTERNAL_BLOCK_SIZE;

    __shared__ real_type data_intern_i[THREAD_BLOCK_SIZE][INTERNAL_BLOCK_SIZE];
    __shared__ real_type data_intern_j[THREAD_BLOCK_SIZE][INTERNAL_BLOCK_SIZE];
    real_type matr[INTERNAL_BLOCK_SIZE][INTERNAL_BLOCK_SIZE] = { 0.0 };
    real_type data_j[INTERNAL_BLOCK_SIZE];

    if (i >= j) {
        i += threadIdx.x * INTERNAL_BLOCK_SIZE;
        const unsigned int ji = j + threadIdx.x * INTERNAL_BLOCK_SIZE;
        j += threadIdx.y * INTERNAL_BLOCK_SIZE;
        for (int vec_index = 0; vec_index < Ncols * Nrows; vec_index += Nrows) {
            {
                #pragma unroll INTERNAL_BLOCK_SIZE
                for (int block_id = 0; block_id < INTERNAL_BLOCK_SIZE; ++block_id) {
                    const int data_index = vec_index + block_id;
                    if (threadIdx.y == block_id)
                        data_intern_i[threadIdx.x][block_id] = data_d[data_index + i];
                    if (threadIdx.y == block_id * 2)
                        data_intern_j[threadIdx.x][block_id] = data_d[data_index + ji];
                }
            }
            __syncthreads();

            #pragma unroll INTERNAL_BLOCK_SIZE
            for (int data_index = 0; data_index < INTERNAL_BLOCK_SIZE; ++data_index) {
                data_j[data_index] = data_intern_j[threadIdx.y][data_index];
            }
            __syncthreads();
            #pragma unroll INTERNAL_BLOCK_SIZE
            for (int x = 0; x < INTERNAL_BLOCK_SIZE; ++x) {
                const real_type data_i = data_intern_i[threadIdx.x][x];
                #pragma unroll INTERNAL_BLOCK_SIZE
                for (int y = 0; y < INTERNAL_BLOCK_SIZE; ++y) {
                    matr[x][y] += data_i * data_j[y];
                }
            }
        }
        #pragma unroll INTERNAL_BLOCK_SIZE
        for (int x = 0; x < INTERNAL_BLOCK_SIZE; ++x) {
            #pragma unroll INTERNAL_BLOCK_SIZE
            for (int y = 0; y < INTERNAL_BLOCK_SIZE; ++y) {
                const real_type temp = (pow(gamma * matr[x][y] + coef0, degree) + QA_cost - q[i + x] - q[j + y]) * add;
                if (i + x > j + y) {
                    atomicAdd(&ret[i + x], temp * d[j + y]);
                    atomicAdd(&ret[j + y], temp * d[i + x]);
                } else if (i + x == j + y) {
                    atomicAdd(&ret[j + y], (temp + cost * add) * d[i + x]);
                }
            }
        }
    }
}

template __global__ void kernel_poly(const float *, float *, const float *, const float *, const float, const float, const int, const int, const int, const int, const int, const float, const float, const float);
template __global__ void kernel_poly(const double *, double *, const double *, const double *, const double, const double, const int, const int, const int, const int, const int, const double, const double, const double);

template <typename real_type>
__global__ void kernel_radial(const real_type *q, real_type *ret, const real_type *d, const real_type *data_d, const real_type QA_cost, const real_type cost, const int Ncols, const int Nrows, const int add, const int start, const int end, const real_type gamma) {
    int i = blockIdx.x * blockDim.x * INTERNAL_BLOCK_SIZE;
    int j = blockIdx.y * blockDim.y * INTERNAL_BLOCK_SIZE;

    __shared__ real_type data_intern_i[THREAD_BLOCK_SIZE][INTERNAL_BLOCK_SIZE];
    __shared__ real_type data_intern_j[THREAD_BLOCK_SIZE][INTERNAL_BLOCK_SIZE];
    real_type matr[INTERNAL_BLOCK_SIZE][INTERNAL_BLOCK_SIZE] = {};
    real_type data_j[INTERNAL_BLOCK_SIZE];

    if (i >= j) {
        i += threadIdx.x * INTERNAL_BLOCK_SIZE;
        const int ji = j + threadIdx.x * INTERNAL_BLOCK_SIZE;
        j += threadIdx.y * INTERNAL_BLOCK_SIZE;
        for (int vec_index = 0; vec_index < Ncols * Nrows; vec_index += Nrows) {
            {
                #pragma unroll(INTERNAL_BLOCK_SIZE)
                for (int block_id = 0; block_id < INTERNAL_BLOCK_SIZE; ++block_id) {
                    const int data_index = vec_index + block_id;
                    if (threadIdx.y == block_id)
                        data_intern_i[threadIdx.x][block_id] = data_d[data_index + i];
                    if (threadIdx.y == block_id * 2)
                        data_intern_j[threadIdx.x][block_id] = data_d[data_index + ji];
                }
            }
            __syncthreads();

            #pragma unroll(INTERNAL_BLOCK_SIZE)
            for (int data_index = 0; data_index < INTERNAL_BLOCK_SIZE; ++data_index) {
                data_j[data_index] = data_intern_j[threadIdx.y][data_index];
            }
            __syncthreads();
            #pragma unroll(INTERNAL_BLOCK_SIZE)
            for (int x = 0; x < INTERNAL_BLOCK_SIZE; ++x) {
                const real_type data_i = data_intern_i[threadIdx.x][x];
                #pragma unroll(INTERNAL_BLOCK_SIZE)
                for (int y = 0; y < INTERNAL_BLOCK_SIZE; ++y) {
                    matr[x][y] += (data_i - data_j[y]) * (data_i - data_j[y]);
                }
            }
        }

        #pragma unroll(INTERNAL_BLOCK_SIZE)
        for (int x = 0; x < INTERNAL_BLOCK_SIZE; ++x) {
            #pragma unroll(INTERNAL_BLOCK_SIZE)
            for (int y = 0; y < INTERNAL_BLOCK_SIZE; ++y) {
                const real_type temp = (exp(-gamma * matr[x][y]) + QA_cost - q[i + x] - q[j + y]) * add;
                if (i + x > j + y) {
                    atomicAdd(&ret[i + x], temp * d[j + y]);
                    atomicAdd(&ret[j + y], temp * d[i + x]);
                } else if (i + x == j + y) {
                    atomicAdd(&ret[j + y], (temp + cost * add) * d[i + x]);
                }
            }
        }
    }
}
template __global__ void kernel_radial(const float *, float *, const float *, const float *, const float, const float, const int, const int, const int, const int, const int, const float);
template __global__ void kernel_radial(const double *, double *, const double *, const double *, const double, const double, const int, const int, const int, const int, const int, const double);

}  // namespace plssvm