#include "hip/hip_runtime.h"
#include "CSVM.hpp"

__global__ void kernel_predict(real_t *data_d, real_t *w, int dim, real_t *out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    real_t temp = 0;
    for(int feature = 0; feature < dim ; ++feature){
        temp += w[feature] * data_d[index * dim + feature];
    }
    if(temp > 0) {
        out[index] = 1;
    }else{
        out[index] = -1;
    }
}

__global__ void kernel_w(real_t* w_d, real_t* data_d, real_t* alpha_d, int count ){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    real_t temp = 0;
    for(int dat = 0; dat < count ; ++dat){
        temp += alpha_d[index] * data_d[dat * count + index];
    }
    w_d[index] = temp;
}


std::vector<real_t> CSVM::predict(real_t *data, int dim , int count){
    real_t *data_d, *out;
    hipMalloc((void **) &data_d, dim * count * sizeof(real_t));
    hipMalloc((void **) &out, count * sizeof(real_t));
    hipMemcpy(data_d, data, dim * count * sizeof(real_t), hipMemcpyHostToDevice);

    kernel_predict<<<((int)count/1024) + 1,  std::min(count, 1024)>>>(data, w_d, dim, out);

    std::vector<real_t> ret(count);
    hipDeviceSynchronize();
    hipMemcpy(&ret[0], out, count * sizeof(real_t), hipMemcpyDeviceToHost);
    hipFree(data_d);
    hipFree(out);

    return ret;
}


void CSVM::load_w(){
    hipMalloc((void **) &w_d, Nfeatures_data * sizeof(real_t));
    real_t *alpha_d;
    hipMalloc((void **) &alpha_d, Nfeatures_data * sizeof(real_t));
    hipMemcpy(alpha_d, &alpha[0], Nfeatures_data* sizeof(real_t), hipMemcpyHostToDevice);

    // TODO:
    // kernel_w<<<((int)Nfeatures_data/1024) + 1,  std::min((int)Nfeatures_data, 1024)>>>(w_d, data_d, alpha_d, Ndatas_data);

    hipDeviceSynchronize();
    hipFree(alpha_d);

}
