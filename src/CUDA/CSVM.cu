#include "hip/hip_runtime.h"
#include "CSVM.hpp"
#include "cuda-kernel.cuh"
#include "svm-kernel.cuh"

int CUDADEVICE = 0;
int count_gpu = 1;

CSVM::CSVM(real_t cost_, real_t epsilon_, unsigned kernel_, real_t degree_, real_t gamma_, real_t coef0_ , bool info_) : cost(cost_), epsilon(epsilon_), kernel(kernel_), degree(degree_), gamma(gamma_), coef0(coef0_), info(info_){
	hipGetDeviceCount(&count_gpu);
	std::cout << "GPUs found: " << count_gpu << std::endl;
}

void CSVM::learn(){
	std::vector<real_t> q;
	std::vector<real_t> b = value;
	#pragma omp parallel sections
	{
	#pragma omp section // generate right side from eguation
		{
			b.pop_back();
			b -= value.back();
		}
		#pragma omp section // generate botom right from A
		{
			QA_cost = kernel_function(data.back(), data.back()) + 1 / cost;
		}
	}
	
	if(info)std::cout << "start CG" << std::endl;
	//solve minimization
	q = CG(b,Nfeatures_data,epsilon);
    alpha.emplace_back(-sum(alpha));
	bias = value.back() - QA_cost * alpha.back() - (q * alpha);
}


real_t CSVM::kernel_function(std::vector<real_t>& xi, std::vector<real_t>& xj){
	switch(kernel){
		case 0: return xi * xj;
		case 1: return std::pow(gamma * (xi*xj) + coef0 ,degree);
		case 2: {real_t temp = 0;
			for(int i = 0; i < xi.size(); ++i){
				temp += (xi-xj)*(xi-xj);
			}
			return exp(-gamma * temp);}
		default: throw std::runtime_error("Can not decide wich kernel!");
	}
	
}


void CSVM::loadDataDevice(){
	hipMallocManaged((void **) &datlast, (Nfeatures_data + CUDABLOCK_SIZE - 1) * sizeof(real_t));
	hipMemset(datlast, 0, Nfeatures_data + CUDABLOCK_SIZE - 1 * sizeof(real_t));
	hipMemcpy(datlast,&data[Ndatas_data - 1][0], Nfeatures_data * sizeof(real_t), hipMemcpyHostToDevice);
	hipMallocManaged((void **) &data_d, Nfeatures_data * (Ndatas_data + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) -1)* sizeof(real_t));

	real_t* col_vec;
	hipHostMalloc((void **) &col_vec, (Ndatas_data + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) -1)  * sizeof(real_t));
	
	#pragma parallel for
	for(size_t col = 0; col < Nfeatures_data ; ++col){
		for(size_t row = 0; row < Ndatas_data - 1; ++row){
			col_vec[row] = data[row][col];
		}
		for(int i = 0 ; i < + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) ; ++i){
			col_vec[i + Ndatas_data - 1] = 0;
		}

		//for(size_t device = 0; device < count_gpu; ++device){
		//	hipSetDevice(device);
			hipMemcpy(data_d + col * (Ndatas_data+ (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) -1), col_vec, (Ndatas_data + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) -1) *sizeof(real_t), hipMemcpyHostToDevice);
		//}
	}
	
	hipHostFree(col_vec);
}

void CSVM::learn(std::string &filename, std::string &output_filename) {
	auto begin_parse = std::chrono::high_resolution_clock::now();
	if(filename.size() > 5 && endsWith(filename, ".arff")){
		arffParser(filename);
	}else{
		libsvmParser(filename);
	}

	auto end_parse = std::chrono::high_resolution_clock::now();
	if(info){std::clog << data.size()<<" Datenpunkte mit Dimension "<< Nfeatures_data  <<" in " << std::chrono::duration_cast<std::chrono::milliseconds>(end_parse - begin_parse).count() << " ms eingelesen" << std::endl << std::endl ;}
	
	hipSetDevice(0);
	loadDataDevice();
	
	auto end_gpu = std::chrono::high_resolution_clock::now();
	
	if(info) std::clog << data.size()<<" Datenpunkte mit Dimension "<< Nfeatures_data <<" in " << std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu - end_parse).count() << " ms auf die Gpu geladen" << std::endl << std::endl ;

	learn();
	auto end_learn = std::chrono::high_resolution_clock::now();
    if(info) std::clog << std::endl << data.size()<<" Datenpunkte mit Dimension "<< Nfeatures_data <<" in " <<std::chrono::duration_cast<std::chrono::milliseconds>(end_learn - end_gpu).count() << " ms gelernt" << std::endl;

	writeModel(output_filename);
	auto end_write = std::chrono::high_resolution_clock::now();
    if(info){std::clog << std::endl << data.size()<<" Datenpunkte mit Dimension "<< Nfeatures_data <<" in " <<std::chrono::duration_cast<std::chrono::milliseconds>(end_write-end_learn).count() << " geschrieben" << std::endl;
    }else if(times){
		std::clog << data.size()<<", "<< Nfeatures_data  <<", " << std::chrono::duration_cast<std::chrono::milliseconds>(end_parse - begin_parse).count() << ", "<< std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu - end_parse).count()<< ", " <<std::chrono::duration_cast<std::chrono::milliseconds>(end_learn - end_gpu).count() << ", " <<std::chrono::duration_cast<std::chrono::milliseconds>(end_write-end_learn).count() << std::endl;
	} 

}


std::vector<real_t>CSVM::CG(const std::vector<real_t> &b,const int imax,  const real_t eps)
{
	const int dept = Ndatas_data - 1;
	dim3 grid((int)dept/(CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) + 1,(int)dept/(CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) + 1);
	dim3 block(CUDABLOCK_SIZE, CUDABLOCK_SIZE);
	real_t *x_d[count_gpu],*r, *d, *r_d[count_gpu], *q_d[count_gpu];
	
	hipHostMalloc((void **) &d, dept *sizeof(real_t));
	hipHostMalloc((void **) &r, dept *sizeof(real_t));

	for(size_t device = 0; device < count_gpu; ++device){
		hipSetDevice(device);
		hipMalloc((void **) &x_d[device], (dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) - 1)*sizeof(real_t));
		init<<< ((int) dept/1024) + 1, std::min(1024, dept)>>>(x_d[device],1,dept);
		init<<< 1,(CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) - 1>>>(x_d[device] + dept, 0 , (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) - 1);
		hipMalloc((void **) &r_d[device], (dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) - 1) *sizeof(real_t));
		init<<< 1,(CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD)>>>(r_d[device] + dept, 0 ,(CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) - 1);
		hipMemcpy(r_d[device],&b[0], dept * sizeof(real_t), hipMemcpyHostToDevice);
		hipMalloc((void **) &q_d[device], (dept +  (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) - 1) * sizeof(real_t));
		hipMemset(q_d[device], 0, (dept +  (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) - 1)  * sizeof(real_t));
		kernel_q<<<((int) dept/CUDABLOCK_SIZE) + 1, std::min((int)CUDABLOCK_SIZE, dept)>>>(q_d[device], data_d, datlast, Nfeatures_data , dept + (CUDABLOCK_SIZE * BLOCKING_SIZE_THREAD) );
	}
	switch(kernel){
		case 0:
			for(size_t device = 0; device < count_gpu; ++device){
				hipSetDevice(device); 
				kernel_linear<<<grid,block>>>(q_d[device], r_d[device], x_d[device] ,data_d, QA_cost, 1/cost, Nfeatures_data , dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD), -1);
			}
			break;
	/*	case 1: 
			kernel_poly<<<grid,block>>>(q_d, r_d, x_d,data_d, QA_cost, 1/cost, Nfeatures_data , dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD), -1, gamma, coef0, degree);
			break;	
		case 2: 
			kernel_radial<<<grid,block>>>(q_d, r_d, x_d,data_d, QA_cost, 1/cost, Nfeatures_data , dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD), -1, gamma);
			break;*/
		default: throw std::runtime_error("Can not decide wich kernel!");
	}

	hipSetDevice(CUDADEVICE); 
	hipDeviceSynchronize();
	hipMemcpy(r, r_d[CUDADEVICE], dept*sizeof(real_t), hipMemcpyDeviceToHost); //TODO: splitten
	
	real_t delta = mult(r, r, dept);	
	const real_t delta0 = delta;
	real_t alpha_cd, beta;
	real_t *Ad, *Ad_d[count_gpu];
	
	hipHostMalloc((void **) &Ad, dept *sizeof(real_t));
	for(size_t device = 0; device < count_gpu; ++device){
		hipSetDevice(device); 
		hipMalloc((void **) &Ad_d[device], (dept +(CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) - 1)  *sizeof(real_t));
	}

	int run;
	for(run = 0; run < imax ; ++run){
		if(info)std::cout << "Start Iteration: " << run << std::endl;
		//Ad = A * d
		for(size_t device = 0; device < count_gpu; ++device){
			hipSetDevice(device);
			hipMemset(Ad_d[device], 0, dept * sizeof(real_t));
			hipMemset(r_d[device] + dept, 0, ((CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) - 1) * sizeof(real_t));
			switch(kernel){
				case 0: 
				kernel_linear<<<grid,block>>>(q_d[device], Ad_d[device], r_d[device], data_d, QA_cost, 1/cost, Nfeatures_data, dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) , 1);
				break;
				// case 1: 
				// kernel_poly<<<grid,block>>>(q_d, Ad_d, r_d, data_d, QA_cost, 1/cost, Nfeatures_data, dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) , 1, gamma, coef0, degree);
				// break;
				// case 2: 
				// kernel_radial<<<grid,block>>>(q_d, Ad_d, r_d, data_d, QA_cost, 1/cost, Nfeatures_data, dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD), 1, gamma);
				// break;
				default: throw std::runtime_error("Can not decide wich kernel!");
			}
		}
		hipSetDevice(CUDADEVICE);
		hipDeviceSynchronize();
		hipMemcpy(d, r_d[CUDADEVICE], dept*sizeof(real_t), hipMemcpyDeviceToHost); //TODO: splitten
		hipMemcpy(Ad, Ad_d[CUDADEVICE], dept*sizeof(real_t), hipMemcpyDeviceToHost); //TODO: splitten

		alpha_cd = delta / mult(d , Ad,  dept);
		for(size_t device = 0; device < count_gpu; ++device){
			hipSetDevice(device);
			add_mult<<< ((int) dept/1024) + 1, std::min(1024, dept)>>>(x_d[device],r_d[device],alpha_cd,dept);
		}
		if(run%50 == 0){
			for(size_t device = 0; device < count_gpu; ++device){
				hipSetDevice(device);
				hipMemcpy(r_d[device], &b[device], dept * sizeof(real_t), hipMemcpyHostToDevice);
				switch(kernel){
					case 0: 
						kernel_linear<<<grid,block>>>(q_d[device], r_d[device], x_d[device], data_d, QA_cost, 1/cost, Nfeatures_data, dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD), -1);
						break;
					/*case 1: 
						kernel_poly<<<grid,block>>>(q_d, r_d, x_d, data_d, QA_cost, 1/cost, Nfeatures_data, dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD), -1, gamma, coef0, degree);
						break;
					case 2: 
						kernel_radial<<<grid,block>>>(q_d, r_d, x_d, data_d, QA_cost, 1/cost, Nfeatures_data, dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) , -1, gamma);
						break;*/
					default: throw std::runtime_error("Can not decide wich kernel!");
				}
			}
			hipSetDevice(CUDADEVICE);
			hipDeviceSynchronize();
			hipMemcpy(r, r_d[CUDADEVICE], dept*sizeof(real_t), hipMemcpyDeviceToHost); //TODO: split
			
		}else{
			for(int index = 0; index < dept; ++index){
				r[index] -= alpha_cd * Ad[index];
			}
		}
		delta = mult(r , r, dept);
		if(delta < eps * eps * delta0) break;
		beta = -mult(r, Ad, dept) / mult(d, Ad, dept);
		add(mult(beta, d, dept),r, d, dept);
		for(size_t device = 0; device < count_gpu; ++device){
			hipSetDevice(device);
			hipMemcpy(r_d[device], d, dept*sizeof(real_t), hipMemcpyHostToDevice);
		}
	}
	if(run == imax) std::clog << "Regard reached maximum number of CG-iterations" << std::endl;
	alpha.resize(dept);
	std::vector<real_t> ret_q(dept);
	hipDeviceSynchronize();
	hipMemcpy(&alpha[CUDADEVICE],x_d[CUDADEVICE], dept * sizeof(real_t), hipMemcpyDeviceToHost);
	hipMemcpy(&ret_q[CUDADEVICE],q_d[CUDADEVICE], dept * sizeof(real_t), hipMemcpyDeviceToHost);
	hipFree(Ad_d);
	hipFree(r_d);
	hipFree(datlast);
	hipHostFree(Ad);
	hipFree(x_d);
	hipHostFree(r);
	hipHostFree(d);
	return ret_q;
}