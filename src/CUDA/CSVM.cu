#include "hip/hip_runtime.h"
#include "CSVM.hpp"
#include "cuda-kernel.cuh"
#include "svm-kernel.cuh"

int CUDADEVICE = 0;

CSVM::CSVM(real_t cost_, real_t epsilon_, unsigned kernel_, real_t degree_, real_t gamma_, real_t coef0_ , bool info_) : cost(cost_), epsilon(epsilon_), kernel(kernel_), degree(degree_), gamma(gamma_), coef0(coef0_), info(info_){}

void CSVM::learn(){
	std::vector<real_t> q;
	std::vector<real_t> b = value;
	#pragma omp parallel sections
	{
	#pragma omp section // generate right side from eguation
		{
			b.pop_back();
			b -= value.back();
		}
		#pragma omp section // generate botom right from A
		{
			QA_cost = kernel_function(data.back(), data.back()) + 1 / cost;
		}
	}
	
	if(info)std::cout << "start CG" << std::endl;
	//solve minimization
	q = CG(b,Nfeatures_data,epsilon);
    alpha.emplace_back(-sum(alpha));
	bias = value.back() - QA_cost * alpha.back() - (q * alpha);
}


real_t CSVM::kernel_function(std::vector<real_t>& xi, std::vector<real_t>& xj){
	switch(kernel){
		case 0: return xi * xj;
		case 1: return std::pow(gamma * (xi*xj) + coef0 ,degree);
		case 2: {real_t temp = 0;
			for(int i = 0; i < xi.size(); ++i){
				temp += (xi-xj)*(xi-xj);
			}
			return exp(-gamma * temp);}
		default: throw std::runtime_error("Can not decide wich kernel!");
	}
	
}


void CSVM::loadDataDevice(){
	hipMalloc((void **) &datlast, (Nfeatures_data + CUDABLOCK_SIZE - 1) * sizeof(real_t));
	hipMemset(datlast, 0, Nfeatures_data + CUDABLOCK_SIZE - 1 * sizeof(real_t));
	hipMemcpy(datlast,&data[Ndatas_data - 1][0], Nfeatures_data * sizeof(real_t), hipMemcpyHostToDevice);
	hipMalloc((void **) &data_d, Nfeatures_data * (Ndatas_data + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) -1)* sizeof(real_t));

	real_t* col_vec;
	hipHostMalloc((void **) &col_vec, (Ndatas_data + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) -1)  * sizeof(real_t));
	
	#pragma parallel for
	for(size_t col = 0; col < Nfeatures_data ; ++col){
		for(size_t row = 0; row < Ndatas_data - 1; ++row){
			col_vec[row] = data[row][col];
		}
		for(int i = 0 ; i < + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) ; ++i){
			col_vec[i + Ndatas_data - 1] = 0;
		}
		hipMemcpy(data_d + col * (Ndatas_data+ (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) -1), col_vec, (Ndatas_data + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) -1) *sizeof(real_t), hipMemcpyHostToDevice);
	}
	hipHostFree(col_vec);
}

void CSVM::learn(std::string &filename, std::string &output_filename) {
	auto begin_parse = std::chrono::high_resolution_clock::now();
	if(filename.size() > 5 && endsWith(filename, ".arff")){
		arffParser(filename);
	}else{
		libsvmParser(filename);
	}

	auto end_parse = std::chrono::high_resolution_clock::now();
	if(info){std::clog << data.size()<<" Datenpunkte mit Dimension "<< Nfeatures_data  <<" in " << std::chrono::duration_cast<std::chrono::milliseconds>(end_parse - begin_parse).count() << " ms eingelesen" << std::endl << std::endl ;}
	
	hipSetDevice(CUDADEVICE);
	loadDataDevice();
	
	auto end_gpu = std::chrono::high_resolution_clock::now();
	
	if(info) std::clog << data.size()<<" Datenpunkte mit Dimension "<< Nfeatures_data <<" in " << std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu - end_parse).count() << " ms auf die Gpu geladen" << std::endl << std::endl ;

	learn();
	auto end_learn = std::chrono::high_resolution_clock::now();
    if(info) std::clog << std::endl << data.size()<<" Datenpunkte mit Dimension "<< Nfeatures_data <<" in " <<std::chrono::duration_cast<std::chrono::milliseconds>(end_learn - end_gpu).count() << " ms gelernt" << std::endl;

	writeModel(output_filename);
	auto end_write = std::chrono::high_resolution_clock::now();
    if(info){std::clog << std::endl << data.size()<<" Datenpunkte mit Dimension "<< Nfeatures_data <<" in " <<std::chrono::duration_cast<std::chrono::milliseconds>(end_write-end_learn).count() << " geschrieben" << std::endl;
    }else if(times){
		std::clog << data.size()<<", "<< Nfeatures_data  <<", " << std::chrono::duration_cast<std::chrono::milliseconds>(end_parse - begin_parse).count() << ", "<< std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu - end_parse).count()<< ", " <<std::chrono::duration_cast<std::chrono::milliseconds>(end_learn - end_gpu).count() << ", " <<std::chrono::duration_cast<std::chrono::milliseconds>(end_write-end_learn).count() << std::endl;
	} 

}


std::vector<real_t>CSVM::CG(const std::vector<real_t> &b,const int imax,  const real_t eps)
{
	const int dept = Ndatas_data - 1;
	dim3 grid((int)dept/(CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) + 1,(int)dept/(CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) + 1);
	dim3 block(CUDABLOCK_SIZE, CUDABLOCK_SIZE);
	real_t *x_d, *r, *d, *r_d, *q_d;

	hipMalloc((void **) &x_d, (dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) - 1)*sizeof(real_t));
	init<<< ((int) dept/1024) + 1, std::min(1024, dept)>>>(x_d,1,dept);
	init<<< 1,(CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) - 1>>>(x_d + dept, 0 , (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) - 1);

	//r = b - (A * x)
	///r = b;
	hipHostMalloc((void **) &r, dept *sizeof(real_t));
	hipMalloc((void **) &r_d, (dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) - 1) *sizeof(real_t));
	init<<< 1,(CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD)>>>(r_d + dept, 0 ,(CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) - 1);
	hipHostMalloc((void **) &d, dept *sizeof(real_t));

	hipMemcpy(r_d,&b[0], dept * sizeof(real_t), hipMemcpyHostToDevice);
	
	hipMalloc((void **) &q_d, (dept +  (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) - 1) * sizeof(real_t));
	hipMemset(q_d, 0, (dept +  (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) - 1)  * sizeof(real_t));
	kernel_q<<<((int) dept/CUDABLOCK_SIZE) + 1, std::min((int)CUDABLOCK_SIZE, dept)>>>(q_d, data_d, datlast, Nfeatures_data , dept + (CUDABLOCK_SIZE * BLOCKING_SIZE_THREAD) );
	switch(kernel){
		case 0: 
			kernel_linear<<<grid,block>>>(q_d, r_d, x_d,data_d, QA_cost, 1/cost, Nfeatures_data , dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD), -1);
			break;
		case 1: 
			kernel_poly<<<grid,block>>>(q_d, r_d, x_d,data_d, QA_cost, 1/cost, Nfeatures_data , dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD), -1, gamma, coef0, degree);
			break;	
		case 2: 
			kernel_radial<<<grid,block>>>(q_d, r_d, x_d,data_d, QA_cost, 1/cost, Nfeatures_data , dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD), -1, gamma);
			break;
		default: throw std::runtime_error("Can not decide wich kernel!");
	}
 
	hipDeviceSynchronize();
	hipMemcpy(r, r_d, dept*sizeof(real_t), hipMemcpyDeviceToHost);
	real_t delta = mult(r, r, dept);	
	const real_t delta0 = delta;
	real_t alpha_cd, beta;
	real_t* Ad, *Ad_d;
	hipHostMalloc((void **) &Ad, dept *sizeof(real_t));
	hipMalloc((void **) &Ad_d, (dept +(CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) - 1)  *sizeof(real_t));

	int run;
	for(run = 0; run < imax ; ++run){
		if(info)std::cout << "Start Iteration: " << run << std::endl;
		//Ad = A * d
		hipMemset(Ad_d, 0, dept * sizeof(real_t));
		hipMemset(r_d + dept, 0, ((CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) - 1) * sizeof(real_t));
	
		switch(kernel){
			case 0: 
				kernel_linear<<<grid,block>>>(q_d, Ad_d, r_d, data_d, QA_cost, 1/cost, Nfeatures_data, dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) , 1);
				break;
			case 1: 
				kernel_poly<<<grid,block>>>(q_d, Ad_d, r_d, data_d, QA_cost, 1/cost, Nfeatures_data, dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) , 1, gamma, coef0, degree);
				break;
			case 2: 
				kernel_radial<<<grid,block>>>(q_d, Ad_d, r_d, data_d, QA_cost, 1/cost, Nfeatures_data, dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD), 1, gamma);
				break;
			default: throw std::runtime_error("Can not decide wich kernel!");
		}
		
		hipDeviceSynchronize();
	
		hipMemcpy(d, r_d, dept*sizeof(real_t), hipMemcpyDeviceToHost);
		hipMemcpy(Ad, Ad_d, dept*sizeof(real_t), hipMemcpyDeviceToHost);
		alpha_cd = delta / mult(d , Ad,  dept);
		add_mult<<< ((int) dept/1024) + 1, std::min(1024, dept)>>>(x_d,r_d,alpha_cd,dept);
		if(run%50 == 0){
			hipMemcpy(r_d, &b[0], dept * sizeof(real_t), hipMemcpyHostToDevice);
			hipDeviceSynchronize();
			switch(kernel){
				case 0: 
					kernel_linear<<<grid,block>>>(q_d, r_d, x_d, data_d, QA_cost, 1/cost, Nfeatures_data, dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD), -1);
					break;
				case 1: 
					kernel_poly<<<grid,block>>>(q_d, r_d, x_d, data_d, QA_cost, 1/cost, Nfeatures_data, dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD), -1, gamma, coef0, degree);
					break;
				case 2: 
					kernel_radial<<<grid,block>>>(q_d, r_d, x_d, data_d, QA_cost, 1/cost, Nfeatures_data, dept + (CUDABLOCK_SIZE*BLOCKING_SIZE_THREAD) , -1, gamma);
					break;
				default: throw std::runtime_error("Can not decide wich kernel!");
			}
			hipDeviceSynchronize();	
			hipMemcpy(r, r_d, dept*sizeof(real_t), hipMemcpyDeviceToHost);
		}else{
			for(int index = 0; index < dept; ++index){
				r[index] -= alpha_cd * Ad[index];
			}
		}
		delta = mult(r , r, dept);
		if(delta < eps * eps * delta0) break;
		beta = -mult(r, Ad, dept) / mult(d, Ad, dept);
		add(mult(beta, d, dept),r, d, dept);
		hipMemcpy(r_d, d, dept*sizeof(real_t), hipMemcpyHostToDevice);
	}
	if(run == imax) std::clog << "Regard reached maximum number of CG-iterations" << std::endl;
	alpha.resize(dept);
	std::vector<real_t> ret_q(dept);
	hipDeviceSynchronize();
	hipMemcpy(&alpha[0],x_d, dept * sizeof(real_t), hipMemcpyDeviceToHost);
	hipMemcpy(&ret_q[0],q_d, dept * sizeof(real_t), hipMemcpyDeviceToHost);
	hipFree(Ad_d);
	hipFree(r_d);
	hipFree(datlast);
	hipHostFree(Ad);
	hipFree(x_d);
	hipHostFree(r);
	hipHostFree(d);
	return ret_q;
}